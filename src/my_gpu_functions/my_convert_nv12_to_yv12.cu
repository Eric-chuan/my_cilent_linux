#include "hip/hip_runtime.h"
#include "cvt.cuh"
#include <cstdio>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

__global__ void convert_nv12_to_yv12(uint8_t * nv12_input, uint8_t * yv12_output)
{
    int ty = threadIdx.x;
    int tx = threadIdx.y;
    int starty = blockIdx.x * blockDim.x;
    int startx = blockIdx.y * blockDim.y;
    //input
    yv12_output[3840 * (2 * (starty + ty)) + (2 * (startx + tx))] = nv12_input[4096 * (2 * (starty + ty)) + (2 * (startx + tx))];
    yv12_output[3840 * (2 * (starty + ty) + 1) + (2 * (startx + tx))] = nv12_input[4096 * (2 * (starty + ty) + 1) + (2 * (startx + tx))];
    yv12_output[3840 * (2 * (starty + ty)) + (2 * (startx + tx) + 1)] = nv12_input[4096 * (2 * (starty + ty)) + (2 * (startx + tx) + 1)];
    yv12_output[3840 * (2 * (starty + ty) + 1) + (2 * (startx + tx) + 1)] = nv12_input[4096 * (2 * (starty + ty) + 1) + (2 * (startx + tx) + 1)];
    int start_uv_input = 4096 * 2160;
    int start_u_output = 3840 * 2160;
    int start_v_output = 3840 * 2160 * 5 / 4;
    yv12_output[start_u_output + 1920 * (starty + ty) + (startx + tx)] = nv12_input[start_uv_input + 4096 * (starty + ty) + (2 * (startx + tx))];
    yv12_output[start_v_output + 1920 * (starty + ty) + (startx + tx)] = nv12_input[start_uv_input + 4096 * (starty + ty) + (2 * (startx + tx) + 1)];
}

void my_convert_nv12_to_yv12(uint8_t * input, uint8_t * output, int dev_id)
{
    hipSetDevice(dev_id);
    dim3 Block(8, 32);
    dim3 Grid(135, 60);
    convert_nv12_to_yv12 <<< Grid, Block >>> (input, output);
    //printf("CVT__called.\n");
    //hipDeviceSynchronize();
}
