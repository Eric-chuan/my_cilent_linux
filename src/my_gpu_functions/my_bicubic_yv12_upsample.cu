#include "hip/hip_runtime.h"
#include "cvt.cuh"
#include <cstdio>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

inline __device__ float fastMax(const float a, const float b)
{
    return (a > b ? a : b);
}

inline __device__ float fastMin(const float a, const float b)
{
    return (a < b ? a : b);
}

inline __device__ float fastTruncate(float value, float min = 0.0f, float max = 1.0f)
{
    return fastMin(max, fastMax(min, value));
}

inline __device__ void cubicSequentialData(int* xIntArray, int* yIntArray, float& dx, float& dy, const float xSource, const float ySource, int scale)
{
    int width = (scale == 2) ? 960 : 480;
    int height = (scale == 2) ? 540 : 270;
    xIntArray[1] = fastTruncate(int(xSource + 1e-5), 0, width - 1);
    xIntArray[0] = fastMax(0, xIntArray[1] - 1);
    xIntArray[2] = fastMin(width - 1, xIntArray[1] + 1);
    xIntArray[3] = fastMin(width - 1, xIntArray[2] + 1);
    dx = xSource - xIntArray[1];

    yIntArray[1] = fastTruncate(int(ySource + 1e-5), 0, height - 1);
    yIntArray[0] = fastMax(0, yIntArray[1] - 1);
    yIntArray[2] = fastMin(height - 1, yIntArray[1] + 1);
    yIntArray[3] = fastMin(height - 1, yIntArray[2] + 1);
    dy = ySource - yIntArray[1];
}

inline __device__ void cubicSequentialDataUV(int* xIntArray, int* yIntArray, float& dx, float& dy, const float xSource, const float ySource, int scale)
{
    int width = (scale == 2) ? 480 : 240;
    int height = (scale == 2) ? 270 : 135;
    xIntArray[1] = fastTruncate(int(xSource + 1e-5), 0, width - 1);
    xIntArray[0] = fastMax(0, xIntArray[1] - 1);
    xIntArray[2] = fastMin(width - 1, xIntArray[1] + 1);
    xIntArray[3] = fastMin(width - 1, xIntArray[2] + 1);
    dx = xSource - xIntArray[1];

    yIntArray[1] = fastTruncate(int(ySource + 1e-5), 0, height - 1);
    yIntArray[0] = fastMax(0, yIntArray[1] - 1);
    yIntArray[2] = fastMin(height - 1, yIntArray[1] + 1);
    yIntArray[3] = fastMin(height - 1, yIntArray[2] + 1);
    dy = ySource - yIntArray[1];
}

inline __device__ float cubicInterpolation(const float v0, const float v1, const float v2, const float v3, const float dx)
{
    return (-0.5f * v0 + 1.5f * v1 - 1.5f * v2 + 0.5f * v3) * dx * dx * dx
            + (v0 - 2.5f * v1 + 2.f * v2 - 0.5f * v3) * dx * dx
            - 0.5f * (v0 - v2) * dx // + (-0.5f * v0 + 0.5f * v2) * dx
            + v1;
}
inline __device__ float bicubicInterpolate(uint8_t* src, float xSource, float ySource, int scale)
{
    int width = (scale == 2) ? 960 : 480;
    int xIntArray[4];
    int yIntArray[4];
    float dx;
    float dy;
    cubicSequentialData(xIntArray, yIntArray, dx, dy, xSource, ySource, scale);

    float temp[4];
    for (int i = 0; i < 4; i++)
    {
        const int offset = yIntArray[i] * width;
        temp[i] = cubicInterpolation((float)(src[offset + xIntArray[0]]), (float)(src[offset + xIntArray[1]]),
                                    (float)(src[offset + xIntArray[2]]), (float)(src[offset + xIntArray[3]]), dx);
    }
    return cubicInterpolation(temp[0], temp[1], temp[2], temp[3], dy);
}

inline __device__ float bicubicInterpolateUV(uint8_t* src, float xSource, float ySource, int scale)
{
    int width = (scale == 2) ? 480 : 240;
    int xIntArray[4];
    int yIntArray[4];
    float dx;
    float dy;
    cubicSequentialDataUV(xIntArray, yIntArray, dx, dy, xSource, ySource, scale);

    float temp[4];
    for (int i = 0; i < 4; i++)
    {
        const int offset = yIntArray[i] * width;
        temp[i] = cubicInterpolation((float)(src[offset + xIntArray[0]]), (float)(src[offset + xIntArray[1]]),
                                    (float)(src[offset + xIntArray[2]]), (float)(src[offset + xIntArray[3]]), dx);
    }
    return cubicInterpolation(temp[0], temp[1], temp[2], temp[3], dy);
}


__global__ void bicubic_upsample_kernel(uint8_t* src,  uint8_t* dst, int scale)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;  //0~1920
    int y = threadIdx.y + blockIdx.y * blockDim.y;  //0~1080
    int xDim = blockDim.x * gridDim.x;  //1920
    int yDim = blockDim.y * gridDim.y;  //1080
    int xDim1 = xDim / 2;
    int yDIm1 = yDim / 2;
    int start_u = xDim * yDim;
    int start_v = start_u * 5 / 4;

    float xSource = (x + 0.5f) / scale - 0.5f;
    float ySource = (y + 0.5f) / scale - 0.5f;
    dst[x + y * xDim] = (uint8_t)(bicubicInterpolate(src, xSource, ySource, scale));
    if ((x % 2 == 0) && (y % 2 == 0)) {
        int x1 = x / 2;
        int y1 = y / 2;
        float xSource1 = (x1 + 0.5f) / scale - 0.5f;
        float ySource1 = (y1 + 0.5f) / scale - 0.5f;
        dst[start_u + x1 + y1 * xDim1] = (uint8_t)(bicubicInterpolateUV(&src[start_u / scale / scale], xSource1, ySource1, scale));
        dst[start_v + x1 + y1 * xDim1] = (uint8_t)(bicubicInterpolateUV(&src[start_v / scale / scale], xSource1, ySource1, scale));
    }
}

void my_bicubic_yv12_upsample(uint8_t* src,  uint8_t* dst, int scale, int dev_id)
{
    hipSetDevice(dev_id);
    dim3 blocks(1920 / 32, 1080 / 20);
    dim3 threads(32, 20);
    bicubic_upsample_kernel <<< blocks, threads >>> (src, dst, scale);
}
