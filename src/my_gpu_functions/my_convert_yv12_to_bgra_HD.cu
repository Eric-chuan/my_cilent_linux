#include "hip/hip_runtime.h"
#include "cvt.cuh"
#include <cstdio>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

__global__ void convert_yv12_to_bgra_HD_kernel(uint8_t * yv12_input, uint8_t * rgba_output)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int Y = yv12_input[y * 1920 + x];
    int V = yv12_input[1920 * 1080 + (x / 2) + 960 * (y / 2)];
    int U = yv12_input[1920 * 1080 * 5 / 4 + (x / 2) + 960 * (y / 2)];
    int R = (298*Y + 411 * V - 57344)>>8;
    int G = (298*Y - 101* U - 211* V+ 34739)>>8;
    int B = (298*Y + 519* U- 71117)>>8;
    int A = 255;
    rgba_output[y * 1920 * 4 + 4 * x] = (uint8_t)((B < 0) ? 0 : ((B > 255) ? 255 : B));
    rgba_output[y * 1920 * 4 + 4 * x + 1] = (uint8_t)((G < 0) ? 0 : ((G > 255) ? 255 : G));
    rgba_output[y * 1920 * 4 + 4 * x + 2] = (uint8_t)((R < 0) ? 0 : ((R > 255) ? 255 : R));
    rgba_output[y * 1920 * 4 + 4 * x + 3] = (uint8_t)A;
}

void my_convert_yv12_to_bgra_HD(uint8_t * input, uint8_t * output, int dev_id)
{
    hipSetDevice(dev_id);
    dim3 Block(32, 20);
    dim3 Grid(1920 / 32, 1080 / 20);
    convert_yv12_to_bgra_HD_kernel <<< Grid, Block >>> (input, output);
    //printf("CVT__called.\n");
    //hipDeviceSynchronize();
}
