#include "hip/hip_runtime.h"
#include "cvt.cuh"
#include <cstdio>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

__global__ void convert_yv12_to_bgra_HD_kernel(uint8_t * yv12_input, uint8_t * rgba_output)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    uint8_t Y = yv12_input[y * 1920 + x];
    uint8_t U = yv12_input[1920 * 1080 + (x / 2) + 960 * (y / 2)];
    uint8_t V = yv12_input[1920 * 1080 * 5 / 4 + (x / 2) + 960 * (y / 2)];
    int B = (int)(Y + (U - 128) + (((U - 128) * 198) >> 8));
    int G = (int)(Y - (((U - 128) * 88) >> 8) - (((V - 128) * 183) >> 8));
    int R = (int)(Y + (V - 128) + (((V - 128) * 103) >> 8));
    int A = 255;
    rgba_output[y * 1920 * 4 + 4 * x] = (uint8_t)((B < 0) ? 0 : ((B > 255) ? 255 : B));
    rgba_output[y * 1920 * 4 + 4 * x + 1] = (uint8_t)((G < 0) ? 0 : ((G > 255) ? 255 : G));
    rgba_output[y * 1920 * 4 + 4 * x + 2] = (uint8_t)((R < 0) ? 0 : ((R > 255) ? 255 : R));
    rgba_output[y * 1920 * 4 + 4 * x + 3] = (uint8_t)A;
}

void my_convert_yv12_to_bgra_HD(uint8_t * input, uint8_t * output, int dev_id)
{
    hipSetDevice(dev_id);
    dim3 Block(32, 20);
    dim3 Grid(1920 / 32, 1080 / 20);
    convert_yv12_to_bgra_HD_kernel <<< Grid, Block >>> (input, output);
    //printf("CVT__called.\n");
    //hipDeviceSynchronize();
}
