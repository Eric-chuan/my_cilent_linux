#include "hip/hip_runtime.h"
#include "cvt.cuh"
#include <cstdio>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

__global__ void convert_nv12_to_bgra(uint8_t * nv12_input, uint8_t * rgba_output)
{
    int y = blockIdx.x * blockDim.x + threadIdx.x;
    int x = blockIdx.y * blockDim.y + threadIdx.y;
    int Y = nv12_input[y * 4096 + x];
    int U = nv12_input[2160 * 4096 + (y / 2) * 4096 + 2 * (x / 2)];
    int V = nv12_input[2160 * 4096 + (y / 2) * 4096 + 2 * (x / 2) + 1];
    int R = (298*Y + 411 * V - 57344)>>8;
    int G = (298*Y - 101* U - 211* V+ 34739)>>8;
    int B = (298*Y + 519* U- 71117)>>8;
    int A = 255;
    rgba_output[y * 3840 * 4 + 4 * x] = (uint8_t)((B < 0) ? 0 : ((B > 255) ? 255 : B));
    rgba_output[y * 3840 * 4 + 4 * x + 1] = (uint8_t)((G < 0) ? 0 : ((G > 255) ? 255 : G));
    rgba_output[y * 3840 * 4 + 4 * x + 2] = (uint8_t)((R < 0) ? 0 : ((R > 255) ? 255 : R));
    rgba_output[y * 3840 * 4 + 4 * x + 3] = (uint8_t)A;
}

void my_convert_nv12_to_bgra(uint8_t * input, uint8_t * output, int dev_id)
{
    hipSetDevice(dev_id);
    dim3 Block(16, 64);
    dim3 Grid(135, 60);
    convert_nv12_to_bgra <<< Grid, Block >>> (input, output);
    //printf("CVT__called.\n");
    //hipDeviceSynchronize();
}
