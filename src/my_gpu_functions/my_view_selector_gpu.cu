#include "hip/hip_runtime.h"
#include "cvt.cuh"
#include <cstdio>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

__global__ void view_selector_kernel(uint8_t* mvstream_data,  uint8_t* selected_view_data, int viewIdx)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;  //0~960
    int y = threadIdx.y + blockIdx.y * blockDim.y;  //0~540
    int xDim = blockDim.x * gridDim.x;  //960
    int yDim = blockDim.y * gridDim.y;  //540
    int xDim1 = xDim / 2;
    int yDim1 = yDim / 2;
    int xDim2 = xDim / 4;
    int yDim2 = yDim / 4;

    int start_u = xDim * yDim;
    int start_v = xDim * yDim * 5 / 4;
    //center
    if (viewIdx == 0) {
        for (int i = 0; i < 2; i++) {
            for (int j = 0; j < 2; j++) {
                selected_view_data[x + i * xDim + (y + j * yDim) * 2 * xDim] = mvstream_data[x + i * xDim + (y + j * yDim) * 4 * xDim];
            }
        selected_view_data[start_u * 4 + x + y * xDim] = mvstream_data[start_u * 16 + x + y * 2 * xDim];
        selected_view_data[start_v * 4 + x + y * xDim] = mvstream_data[start_v * 16 + x + y * 2 * xDim];
        }
    } else if (viewIdx <= 8) {
        if (viewIdx % 2 == 1) {
            selected_view_data[x + y * xDim] = mvstream_data[x + (2 + (viewIdx % 4) / 2) * xDim + (y + viewIdx / 4 * yDim) * 4 * xDim];
            if ((x % 2 == 0)&&(y % 2 == 0)) {
                int x1 = x / 2;  //0~480
                int y1 = y / 2;  //0~270
                selected_view_data[start_u + x1 + y1 * xDim1] = mvstream_data[start_u * 16 + x1 + (2 + (viewIdx % 4) / 2) * xDim1 + (y1 + viewIdx / 4 * yDim1) * xDim1 * 4];
                selected_view_data[start_v + x1 + y1 * xDim1] = mvstream_data[start_v * 16 + x1 + (2 + (viewIdx % 4) / 2) * xDim1 + (y1 + viewIdx / 4 * yDim1) * xDim1 * 4];
            }
        } else {
            selected_view_data[x + y * xDim] = mvstream_data[start_u * 8 + x + (((viewIdx - 1) % 4) / 2) * xDim + (y + (viewIdx - 1) / 4 * yDim) * 4 * xDim];
            if ((x % 2 == 0)&&(y % 2 == 0)) {
                int x1 = x / 2;  //0~480
                int y1 = y / 2;  //0~270
                selected_view_data[start_u + x1 + y1 * xDim1] = mvstream_data[start_u * 18 + x1 + (((viewIdx - 1) % 4) / 2) * xDim1 + (y1 + (viewIdx - 1) / 4 * yDim1) * xDim1 * 4];
                selected_view_data[start_v + x1 + y1 * xDim1] = mvstream_data[start_u * 22 + x1 + (((viewIdx - 1) % 4) / 2) * xDim1 + (y1 + (viewIdx - 1) / 4 * yDim1) * xDim1 * 4];
            }
        }
    } else {
        if (viewIdx % 2 == 1) {
            if ((x % 2 == 0)&&(y % 2 == 0)) {
                int x1 = x / 2;  //0~480
                int y1 = y / 2;  //0~270
                selected_view_data[x1 + y1 * xDim1] = mvstream_data[start_u * 8 + x1 + ((viewIdx / 2) % 4 + 4) * xDim1 + (y1 + (viewIdx / 8 - 1) * yDim1) * 4 * xDim];
            }
            if ((x % 4 == 0)&&(y % 4 == 0)) {
                int x2 = x / 4;
                int y2 = y / 4;
                selected_view_data[start_u / 4 + x2 + y2 * xDim2] = mvstream_data[start_u * 18 + x2 + ((viewIdx / 2) % 4 + 4) * xDim2 + (y2 + (viewIdx / 8 - 1) * yDim2) * xDim1 * 4];
                selected_view_data[start_v / 4 + x2 + y2 * xDim2] = mvstream_data[start_u * 22 + x2 + ((viewIdx / 2) % 4 + 4) * xDim2 + (y2 + (viewIdx / 8 - 1) * yDim2) * xDim1 * 4];
            }
        } else {
            if ((x % 2 == 0)&&(y % 2 == 0)) {
                int x1 = x / 2;  //0~480
                int y1 = y / 2;  //0~270
                selected_view_data[x1 + y1 * xDim1] = mvstream_data[start_u * 8 + x1 + (((viewIdx - 1) / 2) % 4 + 4) * xDim1 + (y1 + ((viewIdx - 1) / 8 + 1) * yDim1) * 4 * xDim];
            }
            if ((x % 4 == 0)&&(y % 4 == 0)) {
                int x2 = x / 4;
                int y2 = y / 4;
                selected_view_data[start_u / 4 + x2 + y2 * xDim2] = mvstream_data[start_u * 18 + x2 + (((viewIdx - 1) / 2) % 4 + 4) * xDim2 + (y2 + ((viewIdx - 1) / 8 + 1) * yDim2) * xDim1 * 4];
                selected_view_data[start_v / 4 + x2 + y2 * xDim2] = mvstream_data[start_u * 22 + x2 + (((viewIdx - 1) / 2) % 4 + 4) * xDim2 + (y2 + ((viewIdx - 1) / 8 + 1) * yDim2) * xDim1 * 4];
            }
        }
    }
}

void view_selector_gpu(uint8_t * input, uint8_t * output, int viewIdx, int dev_id)
{
    hipSetDevice(dev_id);
    dim3 blocks(960 / 32, 540 / 20);
    dim3 threads(32, 20);
    view_selector_kernel <<< blocks, threads >>> (input, output, viewIdx);
}